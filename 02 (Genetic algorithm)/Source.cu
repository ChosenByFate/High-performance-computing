#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <hiprand/hiprand_kernel.h>

#define _CRT_SECURE_NO_WARNINGS
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <malloc.h>
#include <limits>//

#define _Number_Of_Parameters (const char)5

static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct Polynomial
{
	double Coefficients[_Number_Of_Parameters];
	double Error;
	__host__ __device__ Polynomial()
	{
		for (char i = 0; i < _Number_Of_Parameters; ++i)
		{
			//Coefficients[i] = (rand() / double(RAND_MAX) - 0.5) * 0.000'000'000'000'000'1;
			Coefficients[i] = 0.;
		}
	}
	__host__ __device__ ~Polynomial()
	{
		//free(Coefficients);
		//std::cout << "I'm dying" << std::endl;
	}
	__host__ __device__ bool operator<(const Polynomial& other) const
	{
		return this->Error < other.Error;
	}
	//bool operator-(const Polynomial& other) const
	//{
	//	return this->Error - other.Error;
	//}
	static int compare(const void *x1, const void *x2)
	{
		//return ((*(Polynomial*)x1).Error - (*(Polynomial*)x2).Error);
		if (((*(Polynomial*)x1).Error > (*(Polynomial*)x2).Error))
			return 1;
		if (((*(Polynomial*)x1).Error < (*(Polynomial*)x2).Error))
			return -1;
		return 0;
	}
};

double *ReadFromFile(const char *FileName, int *count)
{
	FILE *Stream;
	if ((Stream = fopen(FileName, "r")) == NULL)
	{
		printf("Failed to open file.");
		return nullptr;
	}
	if (*count == NULL)
		fscanf(Stream, "%i", count);
	double *Destination = (double*)malloc(*count * sizeof(double));
	for (int i = 0; i < *count; ++i)
	{
		fscanf(Stream, "%lf", &Destination[i]);
	}
	fclose(Stream);
	return Destination;
}

void WriteToFile(const char *FileName, double *Source, int count = _Number_Of_Parameters, bool saveCount = false, const char *separator = "\t")
{
	FILE *Stream;
	if ((Stream = fopen(FileName, "w")) == NULL)
	{
		printf("Failed to open file.");
		return;
	}
	if (saveCount)
	{
		fprintf(Stream, "%i", count);
		fprintf(Stream, separator);
	}
	for (int i = 0; i < count-1; ++i)
	{
		fprintf(Stream, "%.20lf", Source[i]);
		fprintf(Stream, separator);
	}
	fprintf(Stream, "%.20lf", Source[count - 1]);
	fclose(Stream);
}

__global__ void ClearError(Polynomial *individuals, int numberOfIndividuals)
{
	int individual = blockIdx.x;
	if (individual < numberOfIndividuals)
		individuals[individual].Error = 0;
}

__global__ void Fitness(double *x, double *y, Polynomial *individuals, int numberOfPoints, int numberOfIndividuals)
{
	int individual = blockIdx.x * blockDim.x + threadIdx.x;
	if (individual < numberOfIndividuals)
	{
		double MSE = 0.;
		double approximatingFunction;
		for (int j = 0; j < numberOfPoints; ++j)
		{
			approximatingFunction = 0.;
			for (char k = 0; k < _Number_Of_Parameters; ++k)	// �������� ��������.
			{
				approximatingFunction += individuals[individual].Coefficients[k] * pow(x[j], (double)k);
			}
			MSE += pow(approximatingFunction - y[j], 2);
		}
		individuals[individual].Error = MSE;
	}
}

//������� = numberOfIndividuals - threshold.
__global__ void Crossover(Polynomial *individuals, int numberOfIndividuals, int threshold)
{
	int individual = blockIdx.x + threshold;
	if (individual < numberOfIndividuals)
	{
		for (char i = 0; i < _Number_Of_Parameters; ++i)	//������ - �����.
		{
			individuals[individual].Coefficients[i] = individuals[individual - threshold].Coefficients[i];
		}
	}
}

__global__ void CrossoverNext(Polynomial *individuals, int numberOfIndividuals, int threshold)
{
	int individual = blockIdx.x + threshold;
	if (individual < numberOfIndividuals && !(individual % 2))
	{
		hiprandState state;
		double exchange;
		for (char j = 0; j < _Number_Of_Parameters; ++j)	//�����������.
		{
			hiprand_init((unsigned long long)clock() + individual, 0, 0, &state);
			if ((hiprand_normal(&state) - 0.5f) > 0)	// (2/5 � 3/5) 40% � 60% ����� �� 1 � 2 ���������.
			{
				exchange = individuals[individual].Coefficients[j];
				individuals[individual].Coefficients[j] = individuals[individual + 1].Coefficients[j];
				individuals[individual + 1].Coefficients[j] = exchange;
			}
		}
	}
	//� �����: ������ �������� ������� (����� 1 ������� ��������) - � ������� ��������; ������ - ���������.
}

//������� = threshold - 1.
__global__ void Mutation(Polynomial *individuals, int numberOfIndividuals, int threshold, double mean, double variance)
{
	int individual = blockIdx.x + threshold + 1; 	//First individual is the best. That's why we don't touch it.
	if (individual < numberOfIndividuals)
	{
		hiprandState state;
		double change;
		for (int j = 0; j < _Number_Of_Parameters; ++j)
		{
			hiprand_init((unsigned long long)clock() + individual, 0, 0, &state);
			if ((hiprand_normal(&state) - 0.5f) > 0)	//���� ������� ���� = 50%.
				continue;
			//hiprand_log_normal_double(...)
			hiprand_init((unsigned long long)clock() + individual, 0, 0, &state);
			change = hiprand_normal_double(&state) * variance + mean;
			individuals[individual].Coefficients[j] += change;
		}
	}
}

int main()
{
	srand((int)time(NULL));
	bool dataFromFiles;
	int numberOfPoints = NULL;
	int numberOfIndividuals;
	double mean, variance;
	int numberOfEpochs;
	int numberOfConstantEpochs;
	int currentConstEpoch = 0;
	int threshold;	// ����� ����������� ��������� �� ��� (������) �����.
	double *x = nullptr;
	double *y = nullptr;
	double minimalError = std::numeric_limits<double>::max();
	std::cout << "Points from files (1 - YES, 0 - NO): ";
	std::cin >> dataFromFiles;
	if (dataFromFiles)
	{
		x = ReadFromFile("InputX.txt", &numberOfPoints);
		y = ReadFromFile("InputY.txt", &numberOfPoints);
	}
	else
	{
		std::cout << "Number of points (500 - 1000): ";
		std::cin >> numberOfPoints;
		x = (double*)malloc(numberOfPoints * sizeof(double));
		y = (double*)malloc(numberOfPoints * sizeof(double));
		for (int i = 0; i < numberOfPoints; ++i)
		{
			x[i] = (double)i;
			y[i] = rand() % 41 - 20. + (1000. * i / (i + 500) - i / 5);
		}
		std::cout << "Save points in files? (1 - YES, 0 - NO): ";
		std::cin >> dataFromFiles;
		if (dataFromFiles)
		{
			WriteToFile("InputX.txt", x, numberOfPoints, true, "\n");
			WriteToFile("InputY.txt", y, numberOfPoints, false, "\n");
		}
	}
	double *xGPU;
	double *yGPU;
	HANDLE_ERROR(hipMalloc((void**)&xGPU, numberOfPoints * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&yGPU, numberOfPoints * sizeof(double)));
	HANDLE_ERROR(hipMemcpy(xGPU, x, numberOfPoints * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(yGPU, y, numberOfPoints * sizeof(double), hipMemcpyHostToDevice));

	std::cout << "Number of individuals (1000 - 2000): ";
	std::cin >> numberOfIndividuals;
	std::cout << "Mean of mutation: ";
	std::cin >> mean;
	std::cout << "Variance of mutation: ";
	std::cin >> variance;
	std::cout << "Number of epochs: ";
	std::cin >> numberOfEpochs;
	std::cout << "Number of epochs with constant value of the best fitness: ";
	std::cin >> numberOfConstantEpochs;
	threshold = int(numberOfIndividuals / 2.f + 0.5f);
	
	int threadsPerBlockDim = 32;
	dim3 blockDim(threadsPerBlockDim, 1, 1);
	int blocksPerGridDimX = (int)ceilf(numberOfIndividuals / (float)threadsPerBlockDim);
	dim3 gridDim(blocksPerGridDimX, 1, 1);

	Polynomial *polynomials = (Polynomial*)malloc(numberOfIndividuals * sizeof(Polynomial));
	for (int i = 0; i < numberOfIndividuals; i++)
	{
		polynomials[i] = Polynomial();
	}
	Polynomial *polynomialsGPU;
	HANDLE_ERROR(hipMalloc((void**)&polynomialsGPU, numberOfIndividuals * sizeof(Polynomial)));
	HANDLE_ERROR(hipMemcpy(polynomialsGPU, polynomials, numberOfIndividuals * sizeof(Polynomial), hipMemcpyHostToDevice));
	
	clock_t startTimer, stopTimer;
	startTimer = clock();
	for (int i = 0; i < numberOfEpochs; ++i)
	{
		Fitness<<<gridDim, blockDim>>>(xGPU, yGPU, polynomialsGPU, numberOfPoints, numberOfIndividuals);
		/// ����� ������� ������.
		thrust::sort(thrust::device, polynomialsGPU, polynomialsGPU + numberOfIndividuals);
		HANDLE_ERROR(hipMemcpy(polynomials, polynomialsGPU, numberOfIndividuals * sizeof(Polynomial), hipMemcpyDeviceToHost));
		printf("Epoch %i. Lowest error = %lf\n", i, polynomials[0].Error);
		if (minimalError > polynomials[0].Error)
		{
			minimalError = polynomials[0].Error;
			currentConstEpoch = 0;
		}
		else
		{
			++currentConstEpoch;
			if (currentConstEpoch >= numberOfConstantEpochs)
				break;
		}
		/// ����������� � �������.
		Crossover<<<numberOfIndividuals - threshold, 1>>>(polynomialsGPU, numberOfIndividuals, threshold);
		CrossoverNext<<<numberOfIndividuals - 1 - threshold, 1>>>(polynomialsGPU, numberOfIndividuals, threshold);
		Mutation<<<threshold - 1, 1>>>(polynomialsGPU, numberOfIndividuals, threshold, mean, variance);
		HANDLE_ERROR(hipMemcpy(polynomials, polynomialsGPU, numberOfIndividuals * sizeof(Polynomial), hipMemcpyDeviceToHost));
	}
	stopTimer = clock();
	printf("Time on GPU = %lf seconds.\n", (double)(stopTimer - startTimer) / CLOCKS_PER_SEC);
	HANDLE_ERROR(hipMemcpy(polynomials, polynomialsGPU, numberOfIndividuals * sizeof(Polynomial), hipMemcpyDeviceToHost));
	for (char i = 0; i < _Number_Of_Parameters - 1; ++i)
	{
		printf("%.20lf * x^%i + ", polynomials[0].Coefficients[i], i);
	}
	printf("%.20lf * x^%i\n", polynomials[0].Coefficients[_Number_Of_Parameters - 1], _Number_Of_Parameters - 1);
	printf("blocksPerGridDimX (points) %i.\n", blocksPerGridDimX);
	WriteToFile("Output.txt", polynomials[0].Coefficients);
	HANDLE_ERROR(hipFree(polynomialsGPU));
	HANDLE_ERROR(hipFree(xGPU));
	HANDLE_ERROR(hipFree(yGPU));
	free(polynomials);
	free(x);
	free(y);
//	system("pause");
}